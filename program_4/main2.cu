#include "hip/hip_runtime.h"
#include </Developer/NVIDIA/CUDA-6.5/include/hip/hip_runtime.h>
#include </Developer/NVIDIA/CUDA-6.5/include/hip/hip_runtime.h>
#include </Developer/NVIDIA/CUDA-6.5/include/>
#include <stdio.h>

// Kernel definition
__global__ void MatAdd(float a[N][N], float b[N][N], float c[N][N]) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    c[i][j] = a[i][j] + b[i][j];
}
 
int main() {
    float a[N][N];
    float b[N][N];
    float c[N][N];

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    float (*d_A)[N]; //pointers to arrays of dimension N
    float (*d_B)[N];
    float (*d_C)[N];

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            a[i][j] = i;
            b[i][j] = j;
        }
    }

    // Kernel invocation with one block of N * N * 1 threads
    int numBlocks = 1;
    dim3 threadsPerBlock(N, N);
    MatAdd<<<numBlocks, threadsPerBlock>>>(a, b, c);

    //allocation
    hipMalloc((void**)&d_A, (N*N)*sizeof(float));
    hipMalloc((void**)&d_B, (N*N)*sizeof(float));
    hipMalloc((void**)&d_C, (N*N)*sizeof(float));
}